#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <complex>
#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include <cusp/hip/hip_runtime_api.h>
#include <cusp/dot_product.cuh>
#include <iostream>
#include <stdio.h>

#define default_min_block 256
#define default_min_grid 32

namespace cusp {

// Code is based on "cuda by example: an introduction to general purpose gpu programming."
// I would assume this needs to be licensed / cited but I'm not certain how.
template <typename T>
__global__ void kernel_dot_product(const T *in1, const T *in2, T *out, size_t stride, int N) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ T cache[default_min_block];
  T temp = (T)0;

  while (i < N) {
    temp += in1[stride*i] * in2[i];
    i += blockDim.x * gridDim.x;
  }

  __syncthreads();
  cache[threadIdx.x] = temp;
  int j = blockDim.x / 2;
  
  while (j != 0) {
    if (threadIdx.x < j) cache[threadIdx.x] += cache[j + threadIdx.x];
    __syncthreads();
    j /= 2;
  }

  if (threadIdx.x == 0) out[blockIdx.x] = cache[0];
}

template <>
__global__ void kernel_dot_product<thrust::complex<float>>(
  const thrust::complex<float> *in1, const thrust::complex<float> *in2,
  thrust::complex<float> *out, size_t stride, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ thrust::complex<float> cache[default_min_block];
    thrust::complex<float> temp(0, 0);

    while (i < N) {
      // temp += in1[i] * in2[i];
      temp += in1[stride*i] *  thrust::complex<float>(in2[i].real(), -1.0 * in2[i].imag());
      i += blockDim.x * gridDim.x;
    }

    __syncthreads();
    cache[threadIdx.x] = temp;
    int j = blockDim.x / 2;
    
    while (j != 0) {
      if (threadIdx.x < j) cache[threadIdx.x] += cache[j + threadIdx.x];
      __syncthreads();
      j /= 2;
    }

    if (threadIdx.x == 0) out[blockIdx.x] = cache[0];
}


template <>
__global__ void kernel_dot_product<thrust::complex<double>>(
  const thrust::complex<double> *in1, const thrust::complex<double> *in2,
  thrust::complex<double> *out, size_t stride, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ thrust::complex<double> cache[default_min_block];
    thrust::complex<double> temp(0, 0);

    while (i < N) {
      // temp += in1[i] * in2[i];
      temp += in1[stride*i] *  thrust::complex<double>(in2[i].real(), -1.0 * in2[i].imag());
      i += blockDim.x * gridDim.x;
    }

    __syncthreads();
    cache[threadIdx.x] = temp;
    int j = blockDim.x / 2;
    
    while (j != 0) {
      if (threadIdx.x < j) cache[threadIdx.x] += cache[j + threadIdx.x];
      __syncthreads();
      j /= 2;
    }

    if (threadIdx.x == 0) out[blockIdx.x] = cache[0];
}


// This is a really inneficient way of performing decimation,
// but unfortunately I'm not sure of a better way. A kernel
// implementation allows us to calculate the numeric value of
// our dot product before copying memory back to the host,
// whereas a cpu function requires memory to have already 
// been copied back to the host, which can be annoying when
// working with gnuradio blocks.
template <typename T>
__global__ void kernel_decimate(T * outputs, const int gridSize) {
  for (int i = 1; i < gridSize; i++) {
    outputs[0] += outputs[i];
    outputs[i] = 0;
  }
}

template <>
__global__ void kernel_decimate<thrust::complex<float>>(
  thrust::complex<float> * outputs, const int gridSize) {
  for (int i = 1; i < gridSize; i++) {
    outputs[0] += outputs[i];
    outputs[i] = thrust::complex<float>(0.0, 0.0);
  }
}

template <>
__global__ void kernel_decimate<thrust::complex<double>>(
  thrust::complex<double> * outputs, const int gridSize) {
  for (int i = 1; i < gridSize; i++) {
    outputs[0] += outputs[i];
    outputs[i] = thrust::complex<double>(0.0, 0.0);
  }
}

template <typename T> dot_product<T>::dot_product(size_t stride) : _stride(stride) {

}

template <typename T>
hipError_t dot_product<T>::launch(const std::vector<const void *> &inputs,
                                   T *output, size_t stride, int grid_size, int block_size,
                                   size_t nitems, hipStream_t stream) {

    if (stream) {
      kernel_dot_product<<<grid_size, block_size, 0, stream>>>(
          (const T *)inputs[0],
          (const T *)inputs[1],
          (T *)output, stride, nitems);
      kernel_decimate<<<1, 1, 0, stream>>>(output, grid_size);
    } else {
      kernel_dot_product<<<grid_size, block_size>>>(
          (const T *)inputs[0],
          (const T *)inputs[1],
          (T *)output, stride, nitems);
      kernel_decimate<<<1, 1>>>(output, grid_size);
    }
    return hipPeekAtLastError();
}

template <>
hipError_t dot_product<std::complex<float>>::launch(const std::vector<const void *> &inputs,
                                   std::complex<float> *output, size_t stride, int grid_size, int block_size,
                                   size_t nitems, hipStream_t stream) {

    if (stream) {
      kernel_dot_product<<<grid_size, block_size, 0, stream>>>(
          (const thrust::complex<float> *)inputs[0],
          (const thrust::complex<float> *)inputs[1],
          (thrust::complex<float> *)output, stride, nitems);
      kernel_decimate<<<1, 1, 0, stream>>>(
          (thrust::complex<float> *)output, grid_size);
    } else {
      kernel_dot_product<<<grid_size, block_size>>>(
          (const thrust::complex<float> *)inputs[0],
          (const thrust::complex<float> *)inputs[1],
          (thrust::complex<float> *)output, stride, nitems);
      kernel_decimate<<<1, 1>>>(
          (thrust::complex<float> *)output, grid_size);
    }
    return hipPeekAtLastError();
}


template <>
hipError_t dot_product<std::complex<double>>::launch(const std::vector<const void *> &inputs,
                                   std::complex<double> *output, size_t stride, int grid_size, int block_size,
                                   size_t nitems, hipStream_t stream) {

    if (stream) {
      kernel_dot_product<<<grid_size, block_size, 0, stream>>>(
          (const thrust::complex<double> *)inputs[0],
          (const thrust::complex<double> *)inputs[1],
          (thrust::complex<double> *)output, stride, nitems);
      kernel_decimate<<<1, 1, 0, stream>>>(
          (thrust::complex<double> *)output, grid_size);
    } else {
      kernel_dot_product<<<grid_size, block_size>>>(
          (const thrust::complex<double> *)inputs[0],
          (const thrust::complex<double> *)inputs[1],
          (thrust::complex<double> *)output, stride, nitems);
      kernel_decimate<<<1, 1>>>(
          (thrust::complex<double> *)output, grid_size);
    }
    return hipPeekAtLastError();
}


template <typename T>
hipError_t dot_product<T>::launch(const std::vector<const void *> &inputs,
                                   const std::vector<void *> &outputs,
                                   size_t nitems) {
  return launch(inputs, (T *)outputs[0], _stride, _grid_size, _block_size,
                nitems, _stream);
}

template <typename T> hipError_t dot_product<T>::occupancy(int *minBlock, int *minGrid) {
  *minBlock = default_min_block;
  *minGrid = default_min_grid;
  return hipPeekAtLastError();
}

#define IMPLEMENT_KERNEL(T) template class dot_product<T>;

IMPLEMENT_KERNEL(int8_t)
IMPLEMENT_KERNEL(int16_t)
IMPLEMENT_KERNEL(int32_t)
IMPLEMENT_KERNEL(int64_t)
IMPLEMENT_KERNEL(std::complex<float>);
IMPLEMENT_KERNEL(float)
IMPLEMENT_KERNEL(std::complex<double>);
IMPLEMENT_KERNEL(double)

} // namespace cusp
