#include "hip/hip_runtime.h"
#include <cusp/hip/hip_runtime_api.h>
#include <complex>
#include <limits>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cusp/min.cuh>

#define default_min_block 256
#define default_min_grid 32

namespace cusp {

template <typename T>
__global__ void kernel_min(const T* ins, T* out, T numeric_max,
    int stream_number, int grid_size, int N)
{
    __shared__ T cache[default_min_block];

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int cacheIndex = threadIdx.x;

    T temp = numeric_max;
    while (i < N) {
    	if(ins[i] < temp)
    		temp = ins[i];
        i += blockDim.x * gridDim.x;  
    }
   
    cache[cacheIndex] = temp;

    __syncthreads();

    int ib = blockDim.x / 2;
    while (ib != 0) {
      if(cacheIndex < ib && cache[cacheIndex + ib] < cache[cacheIndex])
        cache[cacheIndex] = cache[cacheIndex + ib]; 

      __syncthreads();

      ib /= 2;
    }
    
    if(cacheIndex == 0) {
        out[blockIdx.x + stream_number * grid_size] = cache[0];
    }
}


template <typename T>
__global__ void decimate_min_single(T * out, int grid_size, int ninputs) {
    T min = out[0];
    for (int i = 0; i < ninputs * grid_size; i++) {
        if (min > out[i]) min = out[i];
    }
    out[0] = min;
}

template <typename T>
__global__ void decimate_min_multiple(T * out, int grid_size, int ninputs) {
    T min = out[0];
    for (int stream_number = 0; stream_number < ninputs; stream_number++) {
        for (int block_index = 0; block_index < grid_size; block_index++) {
            int index = stream_number * grid_size + block_index;
            if (out[index] < min) {
                min = out[index];
            }
        }
        out[stream_number] = min;
    }
}

template <typename T>
hipError_t min<T>::launch(const std::vector<const void *> &inputs,
                                T *output, int ninputs, bool multi_output,
                                int grid_size, int block_size, size_t nitems,
                                hipStream_t stream) {

    T numeric_max = std::numeric_limits<T>::max();

    if (stream) {
        for (int i = 0; i < ninputs; i++) {
            kernel_min<<<grid_size, block_size, 0, stream>>>(
                (const T *)inputs[i],
                (T *)output, numeric_max, i, grid_size, nitems
            );
        }
        if (multi_output) {
            decimate_min_multiple<<<1, 1, 0, stream>>>(
                output, grid_size, ninputs
            );
        } else {
            decimate_min_single<<<1, 1, 0, stream>>>(
                output, grid_size, ninputs
            );
        }
    }
    else {
        for (int i = 0; i < ninputs; i++) {
            kernel_min<<<grid_size, block_size>>>(
                (const T *)inputs[i],
                (T *)output, numeric_max, i, grid_size, nitems
            );
        }
        if (multi_output) {
            decimate_min_multiple<<<1, 1>>>(
                output, grid_size, ninputs
            );
        } else {
            decimate_min_single<<<1, 1>>>(
                output, grid_size, ninputs
            );
        }
    }
    return hipPeekAtLastError();
}

template <typename T>
hipError_t min<T>::launch(const std::vector<const void *> &inputs,
                                const std::vector<void *> &outputs,
                                size_t nitems) {
    return launch(inputs, (T *)outputs[0], _ninputs, _multi_output,
        _grid_size, _block_size, nitems, _stream);
}

template <typename T> hipError_t min<T>::occupancy(int *minBlock, int *minGrid) {
    *minBlock = default_min_block;
    *minGrid = default_min_grid;
    return hipPeekAtLastError();
}

#define IMPLEMENT_KERNEL(T) template class min<T>;

IMPLEMENT_KERNEL(int8_t)
IMPLEMENT_KERNEL(int16_t)
IMPLEMENT_KERNEL(int32_t)
IMPLEMENT_KERNEL(int64_t)
IMPLEMENT_KERNEL(float)
IMPLEMENT_KERNEL(double)

} // namespace cusp