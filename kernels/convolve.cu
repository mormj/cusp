#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cusp/convolve.cuh>
#include "hip/hip_runtime_api.h"

namespace cusp {

extern "C" __global__ void __launch_bounds__(512)
    _cupy_convolve_float32(const float *__restrict__ inp, const int inpW,
                           const float *__restrict__ kernel, const int kerW,
                           const int mode, const bool swapped_inputs,
                           float *__restrict__ out, const int outW);

template <typename T, typename T_TAPS>
convolve<T, T_TAPS>::convolve(const std::vector<T_TAPS> &taps) : _taps(taps) {
  checkCudaErrors(hipMalloc(&_dev_taps, taps.size() * sizeof(T)));
  checkCudaErrors(hipMemcpy(_dev_taps, taps.data(), taps.size() * sizeof(T),
                             hipMemcpyHostToDevice));
};

template <typename T, typename T_TAPS>
hipError_t convolve<T, T_TAPS>::launch(const T *in, T *out, int N,
                                        int grid_size, int block_size,
                                        hipStream_t stream) {
  if (stream) {

    _cupy_convolve_float32<<<grid_size, block_size, 0, stream>>>(
        in, N, _dev_taps, _taps.size(), 2, false, out, N);
  } else {
    _cupy_convolve_float32<<<grid_size, block_size>>>(
        in, N, _dev_taps, _taps.size(), 2, false, out, N);
  }
  return hipPeekAtLastError();
}

template <typename T, typename T_TAPS>
hipError_t convolve<T, T_TAPS>::launch(const std::vector<const void *> inputs,
                                        const std::vector<void *> outputs,
                                        size_t nitems) {
  return launch((const T *)inputs[0], (T *)outputs[0], nitems, _grid_size,
                _block_size, _stream);
}

template <typename T, typename T_TAPS>
hipError_t convolve<T, T_TAPS>::occupancy(int *minBlock, int *minGrid) {
  auto rc = hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                               _cupy_convolve_float32, 0, 0);

  *minBlock =
      std::min(*minBlock, 512); // Convolve kernels are limited to 512 threads

  return rc;
}

#define IMPLEMENT_KERNEL(T, T_TAPS) template class convolve<T, T_TAPS>;

// IMPLEMENT_KERNEL(int8_t)
// IMPLEMENT_KERNEL(int16_t)
// IMPLEMENT_KERNEL(int32_t)
// IMPLEMENT_KERNEL(int64_t)
IMPLEMENT_KERNEL(float, float)
// IMPLEMENT_KERNEL(std::complex<float>)

} // namespace cusp