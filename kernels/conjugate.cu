#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <cusp/conjugate.cuh>

namespace cusp {

template <typename T> __global__ void kernel_conjugate(const T *in, T *out, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    out[i] = thrust::complex<float>(in[i].real(), -1.0 * in[i].imag());
  }
}

template <typename T>
hipError_t conjugate<T>::launch(const T *in, T *out, int N, int grid_size, int block_size,
                  hipStream_t stream) {
  if (stream) {
    kernel_conjugate<<<grid_size, block_size, 0, stream>>>((const thrust::complex<float> *)in, 
                                                           (thrust::complex<float> *)out, N);
  } else {
    kernel_conjugate<<<grid_size, block_size>>>((const thrust::complex<float> *)in, 
                                                (thrust::complex<float> *)out, N);
  }
  return hipPeekAtLastError();
}

template <typename T>
hipError_t conjugate<T>::launch(const std::vector<const void *> inputs,
                  const std::vector<void *> outputs, size_t nitems) {
  return launch((const T*)inputs[0], (T*)outputs[0], nitems, _grid_size, _block_size, _stream);
}

template <typename T> hipError_t conjugate<T>::occupancy(int *minBlock, int *minGrid) {
  return hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                            kernel_conjugate<thrust::complex<float>>, 0, 0);
}

#define IMPLEMENT_KERNEL(T) template class conjugate<T>;

IMPLEMENT_KERNEL(std::complex<float>)

} // namespace cusp
