#include "hip/hip_runtime.h"
#include "../include/cusp/multiply_const.cuh"
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <complex>

namespace cusp {

template <typename T>
__global__ void kernel_multiply_const(const T *in, T *out, T k, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    out[i] = in[i] * k;
  }
}

template <>
__global__ void kernel_multiply_const(const hipFloatComplex* in,
                                      hipFloatComplex* out,
                                      hipFloatComplex k,
                                      int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    out[i].x = k.x * in[i].x - k.y * in[i].y;
    out[i].y = k.x * in[i].y + k.y * in[i].x;
  }
}

template <typename T>
hipError_t multiply_const<T>::launch(const T *in, T *out, T k, int N, int grid_size,
                                 int block_size, hipStream_t stream) {
  if (stream) {
    kernel_multiply_const<<<grid_size, block_size, 0, stream>>>(in, out, k, N);
  } else {
    kernel_multiply_const<<<grid_size, block_size>>>(in, out, k, N);
  }
  return hipPeekAtLastError();
}

template <>
hipError_t multiply_const<std::complex<float>>::launch(const std::complex<float> *in, std::complex<float> *out, std::complex<float> k, int N, int grid_size,
                                 int block_size, hipStream_t stream) {
  if (stream) {
    kernel_multiply_const<hipFloatComplex><<<grid_size, block_size, 0, stream>>>((const hipFloatComplex *)in, (hipFloatComplex *)out, make_hipFloatComplex(k.real(), k.imag()), N);
  } else {
    kernel_multiply_const<hipFloatComplex><<<grid_size, block_size>>>((const hipFloatComplex *)in, (hipFloatComplex *)out, make_hipFloatComplex(k.real(), k.imag()), N);
  }
  return hipPeekAtLastError();
}

template <typename T>
hipError_t multiply_const<T>::launch(const std::vector<const void *> inputs,
                                 const std::vector<void *> outputs,
                                 size_t nitems) {
  return launch((const T *)inputs[0], (T *)outputs[0], _k, nitems, _grid_size,
                _block_size, _stream);
}

template <typename T>
hipError_t multiply_const<T>::occupancy(int *minBlock, int *minGrid) {
  return hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                            kernel_multiply_const<T>, 0, 0);
}

template <>
hipError_t multiply_const<std::complex<float>>::occupancy(int *minBlock, int *minGrid) {
  return hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                            kernel_multiply_const<hipFloatComplex>, 0, 0);
}

#define IMPLEMENT_KERNEL(T) template class multiply_const<T>;

IMPLEMENT_KERNEL(int8_t)
IMPLEMENT_KERNEL(int16_t)
IMPLEMENT_KERNEL(int32_t)
IMPLEMENT_KERNEL(int64_t)
IMPLEMENT_KERNEL(float)
//IMPLEMENT_KERNEL(hipFloatComplex)
IMPLEMENT_KERNEL(std::complex<float>)

} // namespace cusp