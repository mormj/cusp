#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace cusp {

template <typename T> __global__ void kernel_multiply_const(float f, const T *in, T *out, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    out[i] = in[i] * f;
  }
}

template <typename T>
void launch_kernel_multiply_const(float f, const T *in, T *out, int grid_size, int block_size,
                        int N, hipStream_t stream = 0) {

  if (stream) {
    kernel_add_const<<<grid_size, block_size, 0, stream>>>(f, in, out, N);
  } else {
    kernel_add_const<<<grid_size, block_size>>>(f, in, out, N);
  }
}

#define IMPLEMENT_KERNEL(T)                                                    \
  template void launch_kernel_multiply_const(float f, const T *in, T *out, int grid_size,         \
                                   int block_size, int N,                      \
                                   hipStream_t stream);

IMPLEMENT_KERNEL(uint8_t)
IMPLEMENT_KERNEL(uint16_t)
IMPLEMENT_KERNEL(uint32_t)
IMPLEMENT_KERNEL(uint64_t)

} // namespace cusp

