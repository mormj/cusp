#include "hip/hip_runtime.h"
#include <cusp/keep_one_in_n.cuh>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <complex>

namespace cusp {

// integer modulo is slow, consider writing my own modulo function

template <typename T>
__global__ void kernel_keep_one_in_n(const T *in, T *out, int window, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    if (i % window == 0) {
      out[i / window] = in[i];
    }
  }
}


template <>
__global__ void kernel_keep_one_in_n<thrust::complex<float>>(
  const thrust::complex<float> *in, 
  thrust::complex<float> *out,
  int window, int N) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    if (i % window == 0) {
      out[i / window] = in[i];
    }
  }
}

template <>
__global__ void kernel_keep_one_in_n<thrust::complex<double>>(
  const thrust::complex<double> *in, 
  thrust::complex<double> *out,
  int window, int N) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    if (i % window == 0) {
      out[i / window] = in[i];
    }
  }
}


template <typename T>
hipError_t keep_one_in_n<T>::launch(const T *in, T *out, int window, int N, int grid_size,
                                 int block_size, hipStream_t stream) {
  if (stream) {
    kernel_keep_one_in_n<<<grid_size, block_size, 0, stream>>>(in, out, window, N);
  } else {
    kernel_keep_one_in_n<<<grid_size, block_size>>>(in, out, window, N);
  }
  return hipPeekAtLastError();
}



template <>
hipError_t keep_one_in_n<std::complex<float>>::launch(
  const std::complex<float> *in, std::complex<float> *out,
  int window, int N, int grid_size, int block_size,
  hipStream_t stream) {

  if (stream) {
    kernel_keep_one_in_n<<<grid_size, block_size, 0, stream>>>(
      (const thrust::complex<float> *)in,
      (thrust::complex<float> *)out, window, N);
  } else {
    kernel_keep_one_in_n<<<grid_size, block_size>>>(
      (const thrust::complex<float> *)in,
      (thrust::complex<float> *)out, window, N);
  }
  return hipPeekAtLastError();
}


template <>
hipError_t keep_one_in_n<std::complex<double>>::launch(
  const std::complex<double> *in, std::complex<double> *out,
  int window, int N, int grid_size, int block_size,
  hipStream_t stream) {

  if (stream) {
    kernel_keep_one_in_n<<<grid_size, block_size, 0, stream>>>(
      (const thrust::complex<double> *)in,
      (thrust::complex<double> *)out, window, N);
  } else {
    kernel_keep_one_in_n<<<grid_size, block_size>>>(
      (const thrust::complex<double> *)in,
      (thrust::complex<double> *)out, window, N);
  }
  return hipPeekAtLastError();
}





template <typename T>
hipError_t keep_one_in_n<T>::launch(const std::vector<const void *>& inputs,
                                 const std::vector<void *>& outputs,
                                 size_t nitems) {
  return launch((const T *)inputs[0], (T *)outputs[0], _window, nitems, _grid_size,
                _block_size, _stream);
}

template <typename T>
hipError_t keep_one_in_n<T>::occupancy(int *minBlock, int *minGrid) {
  return hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                            kernel_keep_one_in_n<T>, 0, 0);
}

template <>
hipError_t keep_one_in_n<std::complex<float>>::occupancy(int *minBlock, int *minGrid) {
  return hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                            kernel_keep_one_in_n<std::complex<float>>, 0, 0);
}

template <>
hipError_t keep_one_in_n<std::complex<double>>::occupancy(int *minBlock, int *minGrid) {
  return hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                            kernel_keep_one_in_n<std::complex<double>>, 0, 0);
}


#define IMPLEMENT_KERNEL(T) template class keep_one_in_n<T>;

IMPLEMENT_KERNEL(int8_t)
IMPLEMENT_KERNEL(int16_t)
IMPLEMENT_KERNEL(int32_t)
IMPLEMENT_KERNEL(int64_t)
IMPLEMENT_KERNEL(float)
IMPLEMENT_KERNEL(double)
IMPLEMENT_KERNEL(std::complex<float>)
IMPLEMENT_KERNEL(std::complex<double>)

} // namespace cusp