#include "hip/hip_runtime.h"
#include "../include/cusp/nlog10.cuh"
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <complex>

namespace cusp {

template <typename T>
__global__ void kernel_nlog10(const T *in, T *out, T n, T k, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    out[i] = n * (T)log10(float(in[i])) + k;
  }
}

template <typename T>
hipError_t nlog10<T>::launch(const T *in, T *out, T n, T k, int N, int grid_size,
                                 int block_size, hipStream_t stream) {
  if (stream) {
    kernel_nlog10<<<grid_size, block_size, 0, stream>>>(in, out, n, k, N);
  } else {
    kernel_nlog10<<<grid_size, block_size>>>(in, out, n, k, N);
  }
  return hipPeekAtLastError();
}

template <typename T>
hipError_t nlog10<T>::launch(const std::vector<const void *> inputs,
                                 const std::vector<void *> outputs,
                                 size_t nitems) {
  return launch((const T *)inputs[0], (T *)outputs[0], _n, _k, nitems, _grid_size,
                _block_size, _stream);
}

template <typename T>
hipError_t nlog10<T>::occupancy(int *minBlock, int *minGrid) {
  return hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                            kernel_nlog10<T>, 0, 0);
}

#define IMPLEMENT_KERNEL(T) template class nlog10<T>;

IMPLEMENT_KERNEL(int8_t)
IMPLEMENT_KERNEL(int16_t)
IMPLEMENT_KERNEL(int32_t)
IMPLEMENT_KERNEL(int64_t)
IMPLEMENT_KERNEL(float)

} // namespace cusp