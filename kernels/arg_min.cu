#include "hip/hip_runtime.h"
#include <cusp/hip/hip_runtime_api.h>
#include <complex>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cusp/arg_min.cuh>
#include <limits>

#define default_min_block 256
#define default_min_grid 32

namespace cusp {

template <typename T>
__global__ void kernel_arg_min(const T* ins, T* out,
                               int numeric_max, int stream_number,
                               int ninputs, int grid_size, int N)
{
    __shared__ thrust::complex<T> cache[default_min_block];

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int cacheIndex = threadIdx.x;

    T temp = numeric_max;
    while (i < N) {
    	if(ins[i] < temp)
    		temp = ins[i];
        i += blockDim.x * gridDim.x;  
    }
   
    // real component is value, imaginary component is index
    cache[cacheIndex] = thrust::complex<T>(temp, blockDim.x * blockIdx.x + threadIdx.x);

    __syncthreads();

    int ib = blockDim.x / 2;
    while (ib != 0) {
      if(cacheIndex < ib && cache[cacheIndex + ib].real() < cache[cacheIndex].real())
        cache[cacheIndex] = cache[cacheIndex + ib]; 

      __syncthreads();

      ib /= 2;
    }
    
    if(cacheIndex == 0) {
        int index = blockIdx.x + stream_number * grid_size;
        out[index] = cache[0].real();
        out[index + ninputs * grid_size] = cache[0].imag();
    }
}


// First index is index of min value, second index is stream number
template <typename T>
__global__ void decimate_arg_min(T* out, int grid_size, int ninputs) {
    int min_index = 0;
    int min_stream = 0;
    int offset = ninputs * grid_size;
    for (int stream_number = 0; stream_number < ninputs; stream_number++) {
        for (int block_index = 0; block_index < grid_size; block_index++) {
            int index = block_index + stream_number * grid_size;
            if (out[index] < out[min_index]) {
                min_stream = stream_number;
                min_index = index;
            }
        }
    }
    out[0] = (T)out[min_index + offset];
    out[1] = (T)min_stream; 
}

template <typename T>
hipError_t arg_min<T>::launch(const std::vector<const void *> &inputs,
                                T *output, int ninputs, int grid_size, 
                                int block_size, size_t nitems,
                                hipStream_t stream) {

    T numeric_max = std::numeric_limits<T>::max();

    if (stream) {
        for (int i = 0; i < ninputs; i++) {
            kernel_arg_min<<<grid_size, block_size, 0, stream>>>(
                (const T *)inputs[i],
                (T *)output, numeric_max,
                i, ninputs, grid_size, nitems
            );
        }
        hipDeviceSynchronize();
        decimate_arg_min<<<1, 1, 0, stream>>>(output, grid_size, ninputs);
    }
    else {
        for (int i = 0; i < ninputs; i++) {
            kernel_arg_min<<<grid_size, block_size>>>(
                (const T *)inputs[i],
                (T *)output, numeric_max,
                i, ninputs, grid_size, nitems
            );
        }
        hipDeviceSynchronize();
        decimate_arg_min<<<1, 1>>>(output, grid_size, ninputs);
    }
    return hipPeekAtLastError();
}

template <typename T>
hipError_t arg_min<T>::launch(const std::vector<const void *> &inputs,
                                const std::vector<void *> &outputs,
                                size_t nitems) {
    return launch(inputs, (T *)outputs[0], _ninputs, _grid_size,
        _block_size, nitems, _stream);
}

template <typename T> hipError_t arg_min<T>::occupancy(int *minBlock, int *minGrid) {
    *minBlock = default_min_block;
    *minGrid = default_min_grid;
    return hipPeekAtLastError();
}

#define IMPLEMENT_KERNEL(T) template class arg_min<T>;

IMPLEMENT_KERNEL(int8_t)
IMPLEMENT_KERNEL(int16_t)
IMPLEMENT_KERNEL(int32_t)
IMPLEMENT_KERNEL(float)
IMPLEMENT_KERNEL(double)

} // namespace cusp