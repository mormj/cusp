#include "hip/hip_runtime.h"
#include <cusp/window.cuh>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <complex>

namespace cusp {

template <typename T>
__global__ void kernel_window(const T *in, T *out, float * window, int window_length, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    out[i] = in[i] * window[i%window_length];
  }
}

template <typename T>
hipError_t window<T>::launch(const T *in, T *out, float * window, int window_length, int N, int grid_size,
                                 int block_size, hipStream_t stream) {
  if (stream) {
    kernel_window<<<grid_size, block_size, 0, stream>>>(in, out, window, window_length, N);
  } else {
    kernel_window<<<grid_size, block_size>>>(in, out, window, window_length, N);
  }
  return hipPeekAtLastError();
}

template <typename T>
hipError_t window<T>::launch(const std::vector<const void *>& inputs,
                                 const std::vector<void *>& outputs,
                                 size_t nitems) {
  return launch((const T *)inputs[0], (T *)outputs[0], _window, _window_length, nitems, _grid_size,
                _block_size, _stream);
}

template <typename T>
hipError_t window<T>::occupancy(int *minBlock, int *minGrid) {
  return hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                            kernel_window<T>, 0, 0);
}

#define IMPLEMENT_KERNEL(T) template class window<T>;


IMPLEMENT_KERNEL(float)

} // namespace cusp