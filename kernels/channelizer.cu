#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cusp/channelizer.cuh>
#include <cusp/hip/hip_runtime_api.h>

namespace cusp {

extern "C" __global__ void __launch_bounds__(64)
    _cupy_channelizer_8x8_complex64_complex64(
        const int n_chans, const int n_taps, const int n_pts,
        const hipFloatComplex *__restrict__ x,
        const hipFloatComplex *__restrict__ h, hipFloatComplex *__restrict__ y);

extern "C" __global__ void __launch_bounds__(256)
    _cupy_channelizer_16x16_complex64_complex64(
        const int n_chans, const int n_taps, const int n_pts,
        const hipFloatComplex *__restrict__ x,
        const hipFloatComplex *__restrict__ h, hipFloatComplex *__restrict__ y);

extern "C" __global__ void __launch_bounds__(1024)
    _cupy_channelizer_32x32_complex64_complex64(
        const int n_chans, const int n_taps, const int n_pts,
        const hipFloatComplex *__restrict__ x,
        const hipFloatComplex *__restrict__ h, hipFloatComplex *__restrict__ y);

template <typename T>
channelizer<T>::channelizer(const std::vector<T> &taps, const size_t nchans)
    : _taps(taps), _nchans(nchans) {

  std::cout << taps.size() << " " << nchans << std::endl;
  _ntaps = taps.size() / nchans;
  std::cout << _ntaps << std::endl;
  if (_ntaps > 32) {
    throw std::invalid_argument(
        "Number of Taps / Number of Chans must be <= 32");
  }

  checkCudaErrors(hipMalloc(&_dev_taps, taps.size() * sizeof(T)));
  checkCudaErrors(hipMemcpy(_dev_taps, taps.data(), taps.size() * sizeof(T),
                             hipMemcpyHostToDevice));
  occupancy_internal();

  // checkCudaErrors(hipfftPlan1d(&_plan, _nchans, HIPFFT_C2C, 10000000 / _nchans ));
};

template <>
hipError_t channelizer<std::complex<float>>::launch(
    const std::complex<float> *in, std::complex<float> *out, int N,
    int grid_size, int block_size, hipStream_t stream) {

  if (_ntaps <= 8) {
    // std::cout << " launch 1" << std::endl;
    _cupy_channelizer_8x8_complex64_complex64<<<grid_size, block_size, 0,
                                                stream>>>(
        _nchans, _ntaps, N, (const hipFloatComplex *)in,
        (const hipFloatComplex *)_dev_taps, (hipFloatComplex *)out);
  } else if (_ntaps <= 16) {
    // std::cout << " launch 2" << std::endl;
    _cupy_channelizer_16x16_complex64_complex64<<<grid_size, block_size, 0,
                                                  stream>>>(
        _nchans, _ntaps, N, (const hipFloatComplex *)in,
        (const hipFloatComplex *)_dev_taps, (hipFloatComplex *)out);
  } else {
    // std::cout << " launch 3" << std::endl;

    // threadsperblock = (32, 32)
    // blockspergrid = ((n_chans + 31) // 32, _get_numSM() * 2)

    // std::cout << grid_size << " " << block_size << std::endl;
    _cupy_channelizer_32x32_complex64_complex64<<<
        dim3((_nchans + 31) / 32, 40 * 2, 1), dim3(32, 32, 1), 0, stream>>>(
        _nchans, _ntaps, N, (const hipFloatComplex *)in,
        (const hipFloatComplex *)_dev_taps, (hipFloatComplex *)out);
  }

  checkCudaErrors(hipPeekAtLastError());

  // std::cout << "fft with " << _nchans << " / " << N << std::endl;
  checkCudaErrors(hipfftPlan1d(&_plan, _nchans, HIPFFT_C2C, N ));
  checkCudaErrors(hipfftSetStream(_plan, stream));
  checkCudaErrors (hipfftExecC2C(_plan, (hipfftComplex *) out, (hipfftComplex *) out, HIPFFT_FORWARD) );
  _conj_kernel.launch_default_occupancy({out},{out}, N*_nchans);
  hipfftDestroy(_plan);

  return hipPeekAtLastError();
} 

template <typename T>
hipError_t channelizer<T>::launch(const std::vector<const void *> &inputs,
                                   const std::vector<void *> &outputs,
                                   size_t nitems) {
  return launch((const T *)inputs[0], (T *)outputs[0], nitems, _grid_size,
                _block_size, _stream);
}

template <typename T>
hipError_t channelizer<T>::occupancy(int *minBlock, int *minGrid) {
  hipError_t rc;

  if (_ntaps <= 8) {
    rc = hipOccupancyMaxPotentialBlockSize(
        minGrid, minBlock, _cupy_channelizer_8x8_complex64_complex64, 0, 0);
  } else if (_ntaps <= 16) {
    rc = hipOccupancyMaxPotentialBlockSize(
        minGrid, minBlock, _cupy_channelizer_16x16_complex64_complex64, 0, 0);
  } else {
    rc = hipOccupancyMaxPotentialBlockSize(
        minGrid, minBlock, _cupy_channelizer_32x32_complex64_complex64, 0, 0);
  }

  return rc;
}

#define IMPLEMENT_KERNEL(T) template class channelizer<T>;

IMPLEMENT_KERNEL(std::complex<float>)

} // namespace cusp