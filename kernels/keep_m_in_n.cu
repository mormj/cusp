#include "hip/hip_runtime.h"
#include <cusp/keep_m_in_n.cuh>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <complex>

namespace cusp {


// integer modulo is slow, consider writing my own modulo function

template <typename T>
__global__ void kernel_keep_m_in_n(const T *in, T *out, int m, int window, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        if (i % window == 0) {
            int window_number = 0;
            if (i > window - 1) {
                window_number = i / window;
            }
            for (int j = 0; j < m; j++) {
              if (i + j < N) out[j + window_number * m] = in[i + j];
            }
        }
    }
}


template <>
__global__ void kernel_keep_m_in_n<thrust::complex<float>>(
  const thrust::complex<float> *in, 
  thrust::complex<float> *out,
  int m, int window, int N) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
        if (i % window == 0) {
            int window_number = 0;
            if (i > window - 1) {
                window_number = i / window;
            }
            for (int j = 0; j < m; j++) {
              if (i + j < N) out[j + window_number * m] = in[i + j];
            }
        }
    }
}

template <>
__global__ void kernel_keep_m_in_n<thrust::complex<double>>(
    const thrust::complex<double> *in, 
    thrust::complex<double> *out,
    int m, int window, int N) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        if (i % window == 0) {
            int window_number = 0;

            if (i > window - 1) {
                window_number = i / window;
            }

            for (int j = 0; j < m; j++) {
              if (i + j < N) out[j + window_number * m] = in[i + j];
            }
        }
    }
}


template <typename T>
hipError_t keep_m_in_n<T>::launch(const T *in, T *out, int m, int window, int N, int grid_size,
                                 int block_size, hipStream_t stream) {
  if (stream) {
    kernel_keep_m_in_n<<<grid_size, block_size, 0, stream>>>(in, out, m, window, N);
  } else {
    kernel_keep_m_in_n<<<grid_size, block_size>>>(in, out, m, window, N);
  }
  return hipPeekAtLastError();
}



template <>
hipError_t keep_m_in_n<std::complex<float>>::launch(
  const std::complex<float> *in, std::complex<float> *out,
  int m, int window, int N, int grid_size, int block_size,
  hipStream_t stream) {

  if (stream) {
    kernel_keep_m_in_n<<<grid_size, block_size, 0, stream>>>(
      (const thrust::complex<float> *)in,
      (thrust::complex<float> *)out, m, window, N);
  } else {
    kernel_keep_m_in_n<<<grid_size, block_size>>>(
      (const thrust::complex<float> *)in,
      (thrust::complex<float> *)out, m, window, N);
  }
  return hipPeekAtLastError();
}


template <>
hipError_t keep_m_in_n<std::complex<double>>::launch(
  const std::complex<double> *in, std::complex<double> *out,
  int m, int window, int N, int grid_size, int block_size,
  hipStream_t stream) {

  if (stream) {
    kernel_keep_m_in_n<<<grid_size, block_size, 0, stream>>>(
      (const thrust::complex<double> *)in,
      (thrust::complex<double> *)out, m, window, N);
  } else {
    kernel_keep_m_in_n<<<grid_size, block_size>>>(
      (const thrust::complex<double> *)in,
      (thrust::complex<double> *)out, m, window, N);
  }
  return hipPeekAtLastError();
}





template <typename T>
hipError_t keep_m_in_n<T>::launch(const std::vector<const void *>& inputs,
                                 const std::vector<void *>& outputs,
                                 size_t nitems) {
  return launch((const T *)inputs[0], (T *)outputs[0], _m, _window, nitems, _grid_size,
                _block_size, _stream);
}

template <typename T>
hipError_t keep_m_in_n<T>::occupancy(int *minBlock, int *minGrid) {
  return hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                            kernel_keep_m_in_n<T>, 0, 0);
}

template <>
hipError_t keep_m_in_n<std::complex<float>>::occupancy(int *minBlock, int *minGrid) {
  return hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                            kernel_keep_m_in_n<std::complex<float>>, 0, 0);
}

template <>
hipError_t keep_m_in_n<std::complex<double>>::occupancy(int *minBlock, int *minGrid) {
  return hipOccupancyMaxPotentialBlockSize(minGrid, minBlock,
                                            kernel_keep_m_in_n<std::complex<double>>, 0, 0);
}


#define IMPLEMENT_KERNEL(T) template class keep_m_in_n<T>;

IMPLEMENT_KERNEL(int8_t)
IMPLEMENT_KERNEL(int16_t)
IMPLEMENT_KERNEL(int32_t)
IMPLEMENT_KERNEL(int64_t)
IMPLEMENT_KERNEL(float)
IMPLEMENT_KERNEL(double)
IMPLEMENT_KERNEL(std::complex<float>)
IMPLEMENT_KERNEL(std::complex<double>)

} // namespace cusp