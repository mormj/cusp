#include <gtest/gtest.h>
#include <cusp/and.cuh>

using namespace cusp;

template <typename T> 
void run_test(int N, int num_inputs)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = (T)i;
      expected_output_data[i] = (T)(i & i);
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void **dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));

    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::and_bitwise<T> op(num_inputs);

    std::vector<const void *> input_data_pointer_vec(num_inputs);
    for (int i=0; i<num_inputs; i++)
    {
      input_data_pointer_vec[i] = dev_input_data;
    }

    op.launch_default_occupancy({input_data_pointer_vec}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}


TEST(AndKernel, Basic) {
  int N = 1024 * 100;

  run_test<int16_t>(N, 3);
  run_test<int>(N, 3);
}