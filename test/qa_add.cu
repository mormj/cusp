#include <gtest/gtest.h>
#include <complex>
#include <cusp/add.cuh>

using namespace cusp;

template <typename T> 
void run_add_test(int N, T num_inputs)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = (T)i;
      expected_output_data[i] = num_inputs * i;
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void **dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));
  
    std::cout << "ptr: " <<  dev_input_data << std::endl;

    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    int ncopies = N * sizeof(std::complex<float>) / sizeof(T);
    cusp::add<T> op(num_inputs);

    int minGrid, blockSize, gridSize;
    op.occupancy(&blockSize, &minGrid);
    gridSize = (ncopies + blockSize - 1) / blockSize;
    op.set_block_and_grid(blockSize, gridSize);
    op.launch({dev_input_data}, {dev_output_data}, ncopies);
    /*
    int minGrid, minBlock;
    op.occupancy(&minBlock, &minGrid);
    op.set_block_and_grid(minGrid, N / minGrid);

    std::vector<const void *> input_data_pointer_vec(num_inputs);
    for (int i=0; i<num_inputs; i++)
    {
      input_data_pointer_vec[i] = dev_input_data;
    }

    op.launch(input_data_pointer_vec, {dev_output_data}, N);
    */
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}


TEST(Add, Basic) {
  int N = 1024 * 100;

  run_add_test<int16_t>(N, 3);
  // run_add_test<float>(N, 4);
}