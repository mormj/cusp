#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <complex>
#include "../include/cusp/add_const.cuh"

using namespace cusp;


template <typename T> 
void run_test(int N, T k)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = (T)i;
      expected_output_data[i] = i + k;
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::add_const<T> op(k);
    int minGrid, blockSize, gridSize;
    op.occupancy(&blockSize, &minGrid);
    gridSize = (N + blockSize - 1) / blockSize;
    op.set_block_and_grid(blockSize, gridSize);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}

template <> 
void run_test<std::complex<float>>(int N, std::complex<float> k)
{
    std::vector<std::complex<float>> host_input_data(N);
    std::vector<std::complex<float>> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = (std::complex<float>)(float(i), float(i));
      float real = host_input_data[i].real() + k.real();
      float imag = host_input_data[i].imag() + k.imag();
      std::complex<float> temp(real, imag);
      expected_output_data[i] = temp;
    }
    std::vector<std::complex<float>> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(std::complex<float>));
    hipMalloc(&dev_output_data, N * sizeof(std::complex<float>));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(std::complex<float>), hipMemcpyHostToDevice);
  
    cusp::add_const<std::complex<float>> op(k);
    int minGrid, blockSize, gridSize;
    op.occupancy(&blockSize, &minGrid);
    gridSize = (N + blockSize - 1) / blockSize;
    op.set_block_and_grid(blockSize, gridSize);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
  
    for (int i = 0; i < (int)expected_output_data.size(); i++) {

      // Also add a test case to check for imaginary component

      EXPECT_NEAR(expected_output_data[i].real(),
                  host_output_data[i].real(),
                  abs(expected_output_data[i].real() / 10000));

      EXPECT_NEAR(expected_output_data[i].imag(),
                  host_output_data[i].imag(),
                  abs(expected_output_data[i].imag() / 10000));
    }
}


TEST(AddConstKernel, Basic) {
  int N = 1024 * 100;

  run_test<int16_t>(N, 123);
  run_test<float>(N, 456.0001);
  std::complex<float> param(2.0, 2.0);
  run_test<std::complex<float>>(N, param);
}