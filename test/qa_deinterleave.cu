#include <complex>
#include <cusp/deinterleave.cuh>
#include <gtest/gtest.h>
#include <cusp/hip/hip_runtime_api.h>
using namespace cusp;

template <typename T> void run_test(int N, int nstreams, int blocksize) {
  size_t itemsize = sizeof(T);
  std::vector<T> host_input_data(N);
  std::vector<std::vector<T>> expected_output_data(nstreams);
  // std::vector<std::vector<T>> host_output_data(nstreams);
  for (int i = 0; i < nstreams; i++) {
    expected_output_data[i].resize(N / nstreams);
    // host_output_data.resize(N/nstreams);
  }
  for (int i = 0; i < N; i++) {
    host_input_data[i] = (T)i;
    expected_output_data[(i / blocksize) % nstreams]
                        [((i / blocksize) / nstreams) * blocksize +
                         i % blocksize] = (T)i;
  }


  cusp::deinterleave op(nstreams, blocksize, itemsize);
  int minGrid, blockSize, gridSize;
  op.occupancy(&blockSize, &minGrid);
  gridSize = (N + blockSize - 1) / blockSize;
  op.set_block_and_grid(blockSize, gridSize);

  std::vector<void *> output_data_pointer_vec(nstreams);
  for (int i = 0; i < nstreams; i++) {
    void *tmp;
    checkCudaErrors(hipMalloc(&tmp, N * sizeof(T)));
    output_data_pointer_vec[i] = tmp;
  }

  void *dev_input_data;
  hipMalloc(&dev_input_data, N * sizeof(T));

  hipMemcpy(dev_input_data, host_input_data.data(),
  N * sizeof(T), hipMemcpyHostToDevice);

  // virtual hipError_t launch(const std::vector<const void *> &inputs,
  //   const std::vector<void *> &outputs,
  //   size_t nitems) override;
  checkCudaErrors(op.launch({dev_input_data}, output_data_pointer_vec, N * sizeof(T)));

  hipDeviceSynchronize();
  for (int i=0; i<nstreams; i++)
  {
    std::vector<T> host_output_data(N/nstreams);
    hipMemcpy(host_output_data.data(), output_data_pointer_vec[i], (N/nstreams) * sizeof(T),
                hipMemcpyDeviceToHost);

    EXPECT_EQ(expected_output_data[i], host_output_data);
  }
}

TEST(deinterleave, Basic) {
  int N = 30;

  run_test<int16_t>(3*(N / 3), 3, 2);
  run_test<float>(5*(N / 5), 5, 1);
}