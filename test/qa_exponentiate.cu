#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <complex>
#include <cusp/exponentiate.cuh>
#include <cmath>
#include <hip/hip_complex.h>

using namespace cusp;


template <> 
void run_test<hipFloatComplex>(int N, float e)
{
    std::vector<hipFloatComplex> host_input_data(N);
    std::vector<hipFloatComplex> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = make_hipFloatComplex(float(i), float(2 * i));

      float theta = atan2(h_a[i].y, h_a[i].x);
      float mag = sqrtf(powf(h_a[i].x, 2) + powf(h_a[i].y, 2));

      float x = powf(mag, e) * cos(theta * e); 
      float y = powf(mag, e) * sin(theta * e);

      expected_output_data[i] = make_hipFloatComplex(x, y);
    }
    std::vector<hipFloatComplex> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(hipFloatComplex));
    hipMalloc(&dev_output_data, N * sizeof(hipFloatComplex));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  
    cusp::exponentiate<hipFloatComplex> op(e);
    int minGrid, minBlock;
    op.occupancy(&minBlock, &minGrid);
    op.set_block_and_grid(minGrid, N / minGrid);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}

template <typename T> 
void run_test(int N, float e)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = i;
      expected_output_data[i] = (T)powf(float(i), e);
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::exponentiate<T> op(e);
    int minGrid, minBlock;
    op.occupancy(&minBlock, &minGrid);
    op.set_block_and_grid(minGrid, N / minGrid);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}


TEST(ComplexToMagKernel, Basic) {
  int N = 1024 * 100;
  float e = 2.0f;

  run_test<hipFloatComplex>(N, e);
  run_test<float>(N, e);
}
