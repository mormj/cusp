#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <complex>
#include "../include/cusp/divide.cuh"

using namespace cusp;

template <typename T> 
void run_test(int N, int num_inputs)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);

    for (int i = 0; i < N; i++) {
      host_input_data[i] = (T)(i + 1);
      T out = host_input_data[i];
      for (int j = 0; j < num_inputs - 1; j++) {
        out /= host_input_data[i];
      }
      expected_output_data[i] = out;
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void **dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));

    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::divide<T> op(num_inputs);
    int minGrid, blockSize, gridSize;
    op.occupancy(&blockSize, &minGrid);
    gridSize = (N + blockSize - 1) / blockSize;
    op.set_block_and_grid(blockSize, gridSize);

    std::vector<const void *> input_data_pointer_vec(num_inputs);
    for (int i=0; i<num_inputs; i++)
    {
      input_data_pointer_vec[i] = dev_input_data;
    }

    op.launch(input_data_pointer_vec, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}

template <> 
void run_test<std::complex<float>>(int N, int num_inputs)
{
    std::vector<std::complex<float>> host_input_data(N);
    std::vector<std::complex<float>> expected_output_data(N);

    for (int i = 0; i < N; i++) {
      host_input_data[i] = std::complex<float>(float(i + 1), float(i + 1));
      std::complex<float> out = host_input_data[i];
      for (int j = 0; j < num_inputs - 1; j++) {
        out /= host_input_data[i];
      }
      expected_output_data[i] = out;
    }
    std::vector<std::complex<float>> host_output_data(N);
  
    void *dev_input_data;
    void **dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(std::complex<float>));
    hipMalloc(&dev_output_data, N * sizeof(std::complex<float>));

    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(std::complex<float>), hipMemcpyHostToDevice);
  
    cusp::divide<std::complex<float>> op(num_inputs);
    int minGrid, blockSize, gridSize;
    op.occupancy(&blockSize, &minGrid);
    gridSize = (N + blockSize - 1) / blockSize;
    op.set_block_and_grid(blockSize, gridSize);

    std::vector<const void *> input_data_pointer_vec(num_inputs);
    for (int i=0; i<num_inputs; i++)
    {
      input_data_pointer_vec[i] = dev_input_data;
    }

    op.launch(input_data_pointer_vec, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
  
    //EXPECT_EQ(expected_output_data, host_output_data);

    for (int i = 0; i < (int)expected_output_data.size(); i++) {

      // Also add a test case to check for imaginary component

      EXPECT_NEAR(expected_output_data[i].real(),
                  host_output_data[i].real(),
                  abs(expected_output_data[i].real() / 10000));

      EXPECT_NEAR(expected_output_data[i].imag(),
                  host_output_data[i].imag(),
                  abs(expected_output_data[i].imag() / 10000));
    }
}

TEST(DivideKernel, Basic) {
  int N = 10240;

  run_test<int16_t>(N, 3);
  run_test<float>(N, 3);
  run_test<std::complex<float>>(N, 3);
}