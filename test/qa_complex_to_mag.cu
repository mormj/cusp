#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <complex>
#include <cusp/complex_to_mag.cuh>
#include <cmath>

using namespace cusp;

void run_test(int N)
{
    std::vector<std::complex<float>> host_input_data(N);
    std::vector<float> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = std::complex<float>(float(i), float(i * 2));
      expected_output_data[i] = sqrtf(powf(host_input_data[i].real(), 2) + powf(host_input_data[i].imag(), 2));
    }
    std::vector<float> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(std::complex<float>));
    hipMalloc(&dev_output_data, N * sizeof(float));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(std::complex<float>), hipMemcpyHostToDevice);
  
    cusp::complex_to_mag op;
    op.launch_default_occupancy({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(float), hipMemcpyDeviceToHost);
  
    // EXPECT_EQ(expected_output_data, host_output_data);
    for (size_t i = 0; i < expected_output_data.size(); i++) {
      EXPECT_NEAR(expected_output_data[i],
                  host_output_data[i],
                  expected_output_data[i] / 10000);
    }
}


TEST(ComplexToMagKernel, Basic) {
  int N = 1024 * 100;

  run_test(N);
}