#include <gtest/gtest.h>
#include "../include/cusp/not.cuh"

using namespace cusp;


template <typename T> 
void run_test(int N)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = (T)i;
      expected_output_data[i] = ~i;
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::not_bitwise<T> op;
    int minGrid, blockSize, gridSize;
    op.occupancy(&blockSize, &minGrid);
    gridSize = (N + blockSize - 1) / blockSize;
    op.set_block_and_grid(blockSize, gridSize);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}


TEST(NotKernel, Basic) {
  int N = 1024 * 100;

  run_test<int16_t>(N);
  run_test<int>(N);
}