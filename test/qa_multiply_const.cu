#include <gtest/gtest.h>
#include <complex>
#include <cusp/multiply_const.cuh>
#include <hip/hip_complex.h>

using namespace cusp;

template <> 
void run_test<hipFloatComplex>(int N, float k)
{
    std::vector<hipFloatComplex> host_input_data(N);
    std::vector<hipFloatComplex> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = hipFloatComplex(float(i), float(i));
      expected_output_data[i] = hipFloatComplex(float(i) * k, float(i) * k);
    }
    std::vector<hipFloatComplex> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(hipFloatComplex));
    hipMalloc(&dev_output_data, N * sizeof(hipFloatComplex));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  
    cusp::multiply_const<hipFloatComplex> op(k);
    int minGrid, minBlock;
    op.occupancy(&minBlock, &minGrid);
    op.set_block_and_grid(minGrid, N / minGrid);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}


template <typename T> 
void run_test(int N, T k)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = (T)i;
      expected_output_data[i] = i * k;
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::multiply_const<T> op(k);
    int minGrid, minBlock;
    op.occupancy(&minBlock, &minGrid);
    op.set_block_and_grid(minGrid, N / minGrid);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}


TEST(MultiplyConstKernel, Basic) {
  int N = 1024 * 100;
  float e = 2.0f;

  run_test<int16_t>(N, 2);
  run_test<float>(N, 3.0);
  run_test<hipFloatComplex>(N, e);
}