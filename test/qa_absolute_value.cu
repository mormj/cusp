#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <cusp/absolute_value.cuh>

using namespace cusp;

template <typename T> 
void run_test(int N)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = T(i);
      expected_output_data[i] = (T)i;
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::absolute_value<T> op;
    int minGrid, blockSize, gridSize;
    op.occupancy(&blockSize, &minGrid);
    gridSize = (N + blockSize - 1) / blockSize;
    op.set_block_and_grid(blockSize, gridSize);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}

template <> 
void run_test<std::complex<float>>(int N)
{
    std::vector<std::complex<float>> host_input_data(N);
    std::vector<std::complex<float>> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = std::complex<float>(float(i), float(i * 2));
      float mag = sqrtf(powf(host_input_data[i].real(), 2) + powf(host_input_data[i].imag(), 2));
      expected_output_data[i] = std::complex<float>(mag, 0.0f);
    }
    std::vector<std::complex<float>> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(std::complex<float>));
    hipMalloc(&dev_output_data, N * sizeof(std::complex<float>));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(std::complex<float>), hipMemcpyHostToDevice);
  
    cusp::absolute_value<std::complex<float>> op;
    int minGrid, blockSize, gridSize;
    op.occupancy(&blockSize, &minGrid);
    gridSize = (N + blockSize - 1) / blockSize;
    op.set_block_and_grid(blockSize, gridSize);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
  
    //EXPECT_EQ(expected_output_data, host_output_data);
    for (int i = 0; i < (int)expected_output_data.size(); i++) {
      EXPECT_NEAR(expected_output_data[i].real(),
                  host_output_data[i].real(),
                  expected_output_data[i].real() / 10000);
    }
}


TEST(AbsKernel, Basic) {
  int N = 1024 * 100;

  run_test<int32_t>(N);
  run_test<std::complex<float>>(N);
}