#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <complex>
#include <cusp/absolute_value.cuh>
#include <cmath>
#include <hip/hip_complex.h>

using namespace cusp;


template <typename T> 
void run_test(int N)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = T(-1 * i);
      expected_output_data[i] = (T)(i));
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::absolute_value<T> op();
    int minGrid, minBlock;
    op.occupancy(&minBlock, &minGrid);
    op.set_block_and_grid(minGrid, N / minGrid);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}

template <> 
void run_test<hipFloatComplex>(int N)
{
    std::vector<hipFloatComplex> host_input_data(N);
    std::vector<hipFloatComplex> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = make_hipFloatComplex(float(i), float(i * 2));
      float mag = sqrtf(powf(in[i].x, 2) + powf(in[i].y, 2));
      expected_output_data[i] = make_hipFloatComplex(mag, 0.0f);
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::absolute_value<T> op();
    int minGrid, minBlock;
    op.occupancy(&minBlock, &minGrid);
    op.set_block_and_grid(minGrid, N / minGrid);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}


TEST(AbsKernel, Basic) {
  int N = 1024 * 100;

  run_test<int>(N);
  run_test<float>(N);
  run_test<hipFloatComplex>(N);
}