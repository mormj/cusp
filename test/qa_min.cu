#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <complex>
#include <cusp/min.cuh>
#include <algorithm>

using namespace cusp;

template <typename T> 
void run_test(int N, int num_inputs, bool multiple_outputs)
{
    // int grid_size = int((N + 256 - 1) / 256);
    // int output_size = grid_size * num_inputs;

    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);

    for (int i = 0; i < N; i++) {host_input_data[i] = (T)(i + 5);
    }

    expected_output_data[0] = *std::min_element(
         host_input_data.begin(), host_input_data.end());
    
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));

    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::min<T> op(num_inputs, multiple_outputs);

    std::vector<const void *> input_data_pointer_vec(num_inputs);
    for (int i=0; i<num_inputs; i++)
    {
      input_data_pointer_vec[i] = dev_input_data;
    }

    op.launch_default_occupancy({input_data_pointer_vec}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);

    // std::cout << host_output_data[0] << std::endl;

    // for (int i = 0; i < output_size; i++) {
    //   std::cout << host_output_data[i] << " " << i << std::endl;
    // }

    if (multiple_outputs) {
      for (int i = 0; i < num_inputs; i++) {
        EXPECT_EQ(expected_output_data[0], host_output_data[i]);
      }
    } else {
      EXPECT_EQ(expected_output_data[0], host_output_data[0]);
    }
}

TEST(MinKernel, Basic) {
  // In order for min to function properly, N must
  // be a multiple of 256. Hence, when we invoke our
  // kernel, we will internally convert to the nearest
  // multiple of 256 that is greater than N. Then, during
  // our analysis, any values after our orignal array
  // size will be ignored.
  int N = 1024 * 100;

  run_test<int32_t>(N, 1, false);
  run_test<int64_t>(N, 2, true);
  run_test<float>(N, 3, false);
  run_test<double>(N, 4, true);
}