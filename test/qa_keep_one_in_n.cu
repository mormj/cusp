#include <gtest/gtest.h>
#include <complex>
#include <cusp/keep_one_in_n.cuh>
#include <cmath>

using namespace cusp;

template <typename T> 
void run_test(int N, int window)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = (T)i;
      if (i % window == 0) {
        expected_output_data[ i / window ] = host_input_data[i];
      }
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::keep_one_in_n<T> op(window);
    op.launch_default_occupancy({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}

template <> 
void run_test<std::complex<float>>(int N, int window)
{
    std::vector<std::complex<float>> host_input_data(N);
    std::vector<std::complex<float>> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = std::complex<float>(float(i), float(i));
      if (i % window == 0) {
        expected_output_data[ i / window ] = host_input_data[i];
      }
    }
    std::vector<std::complex<float>> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(std::complex<float>));
    hipMalloc(&dev_output_data, N * sizeof(std::complex<float>));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(std::complex<float>), hipMemcpyHostToDevice);
  
    cusp::keep_one_in_n<std::complex<float>> op(window);
    op.launch_default_occupancy({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}


TEST(KeepOneInNKernel, Basic) {
  int N = 1024 * 100;
  run_test<int32_t>(N, 5);
  run_test<float>(N, 15);
  run_test<std::complex<float>>(N, 30);
}