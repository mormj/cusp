#include <gtest/gtest.h>
#include <complex>
#include <cusp/conjugate.cuh>
#include <cmath>
#include <hip/hip_complex.h>

using namespace cusp;


template <typename T> 
void run_test(int N)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = hipFloatComplex(float(i), float(-i));
      expected_output_data[i] = hipFloatComplex(float(i), float(i));
    }
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::conjugate<T> op();
    int minGrid, minBlock;
    op.occupancy(&minBlock, &minGrid);
    op.set_block_and_grid(minGrid, N / minGrid);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(expected_output_data, host_output_data);
}


TEST(ConjugateKernel, Basic) {
  int N = 1024 * 100;

  run_test<hipFloatComplex>(N);
}