#include <gtest/gtest.h>
#include <complex>
#include <cusp/arg_min.cuh>
#include <algorithm>

using namespace cusp;

template <typename T> 
void run_test(int N, int num_inputs)
{
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);

    for (int i = 0; i < N; i++) {
      host_input_data[i] = (T)(N - i + 2);
    }
    expected_output_data[0] = std::min_element(host_input_data.begin(),
                                               host_input_data.end()) - 
                                               host_input_data.begin();

    
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void **dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));

    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::arg_min<T> op(num_inputs);

    std::vector<const void *> input_data_pointer_vec(num_inputs);
    for (int i=0; i<num_inputs; i++)
    {
      input_data_pointer_vec[i] = dev_input_data;
    }

    op.launch_default_occupancy({input_data_pointer_vec}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);

    EXPECT_EQ(expected_output_data[0], host_output_data[0]);
    EXPECT_EQ(true, host_output_data[1] < num_inputs);

}

TEST(ArgMinKernel, Basic) {
  int N = 1024 * 100;

  run_test<int32_t>(N, 1);
  run_test<float>(N, 2);
  run_test<int16_t>(N, 3);
  run_test<double>(N, 4);
}