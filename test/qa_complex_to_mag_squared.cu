#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <complex>
#include <cusp/complex_to_mag_squared.cuh>
#include <cmath>

using namespace cusp;

template <typename T> 
void run_test(int N)
{
    std::vector<std::complex<float>> host_input_data(N);
    std::vector<std::complex<float>> expected_output_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = std::complex<float>(float(i), float(i * 2));
      float mag = powf(host_input_data[i].real(), 2) + powf(host_input_data[i].imag(), 2);
      expected_output_data[i] = std::complex<float>(mag, 0.0f);
    }
    std::vector<std::complex<float>> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(std::complex<float>));
    hipMalloc(&dev_output_data, N * sizeof(std::complex<float>));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(std::complex<float>), hipMemcpyHostToDevice);
  
    cusp::complex_to_mag_squared<std::complex<float>> op;
    int minGrid, blockSize, gridSize;
    op.occupancy(&blockSize, &minGrid);
    gridSize = (N + blockSize - 1) / blockSize;
    op.set_block_and_grid(blockSize, gridSize);
    op.launch({dev_input_data}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
  
    //EXPECT_EQ(expected_output_data, host_output_data);
    for (int i = 0; i < (int)expected_output_data.size(); i++) {
      EXPECT_NEAR(expected_output_data[i].real(),
                  host_output_data[i].real(),
                  expected_output_data[i].real() / 10000);
    }
}


TEST(ComplexToMagSquaredKernel, Basic) {
  int N = 1024 * 100;

  run_test<std::complex<float>>(N);
}