#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <complex>
#include <cusp/max.cuh>
#include <algorithm>

using namespace cusp;

template <typename T> 
void run_test(int N, int num_inputs, bool multiple_outputs)
{
    // int grid_size = int((N + 256 - 1) / 256);
    // int output_size = grid_size * num_inputs;
    std::vector<T> host_input_data(N);
    std::vector<T> expected_output_data(N);

    for (int i = 0; i < N; i++) {
      host_input_data[i] = (T)i;
    }
    expected_output_data[0] = *std::max_element(
        host_input_data.begin(), host_input_data.end());
    
    std::vector<T> host_output_data(N);
  
    void *dev_input_data;
    void **dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(T));
    hipMalloc(&dev_output_data, N * sizeof(T));

    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(T), hipMemcpyHostToDevice);
  
    cusp::max<T> op(num_inputs, multiple_outputs);

    std::vector<const void *> input_data_pointer_vec(num_inputs);
    for (int i=0; i<num_inputs; i++)
    {
      input_data_pointer_vec[i] = dev_input_data;
    }

    op.launch_default_occupancy({input_data_pointer_vec}, {dev_output_data}, N);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(T), hipMemcpyDeviceToHost);

    // std::cout << "from test case" << std::endl;

    // for (int i = 0; i < N; i++) {
    //   if (host_output_data[i] != (T)0) {
    //     std::cout << host_output_data[i] << " " << i << std::endl;
    //   }
    // }

    if (multiple_outputs) {
      for (int i = 0; i < num_inputs; i++) {
        EXPECT_EQ(expected_output_data[0], host_output_data[i]);
      }
    } else {
      EXPECT_EQ(expected_output_data[0], host_output_data[0]);
    }
}

TEST(MaxKernel, Basic) {
  int N = 1024 * 100;

  run_test<int16_t>(N, 3, true);
  run_test<int32_t>(N, 3, false);
  run_test<float>(N, 3, true);
  run_test<double>(N, 3, false);
}